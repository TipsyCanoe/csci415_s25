#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>

#define BLOCK_SIZE 16

__global__ void jacobi_compute_kernel(float *current, float *next, int n, bool *fixed) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = y * n + x;

    if (x > 0 && x < n - 1 && y > 0 && y < n - 1 && !fixed[idx]) {
        next[idx] = 0.25f * (current[(y - 1) * n + x] +
                              current[(y + 1) * n + x] +
                              current[y * n + (x - 1)] +
                              current[y * n + (x + 1)]);
    } else if (fixed[idx] || x == 0 || x == n - 1 || y == 0 || y == n - 1) {
        // Keep fixed points unchanged
        next[idx] = current[idx];
    }
}

__global__ void reduction_kernel(float *current, float *next, float *delta, int n) {
    __shared__ float shared_diff[BLOCK_SIZE * BLOCK_SIZE];

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = threadIdx.y * blockDim.x + threadIdx.x;

    shared_diff[idx] = 0.0f;
    if (x < n && y < n) {
        float diff = fabs(next[y * n + x] - current[y * n + x]);
        shared_diff[idx] = diff;
    }
    __syncthreads();

    for (int stride = (BLOCK_SIZE * BLOCK_SIZE) / 2; stride > 0; stride >>= 1) {
        if (idx < stride) {
            shared_diff[idx] = fmaxf(shared_diff[idx], shared_diff[idx + stride]);
        }
        __syncthreads();
    }

    if (idx == 0) {
        atomicMax((int *)delta, __float_as_int(shared_diff[0]));
    }
}

// Host functions
void initialize_grid(float *grid, bool *fixed, int n) {
    for (int i = 0; i < n * n; ++i) {
        grid[i] = 0.0f;
        fixed[i] = false;
    }
}

void read_input_file(const char *filename, int *n, float *tolerance, float *grid, bool *fixed) {
    FILE *file = fopen(filename, "r");
    if (!file) {
        perror("Error opening file");
        exit(EXIT_FAILURE);
    }

    if (fscanf(file, "%d %f", n, tolerance) != 2) {
        fprintf(stderr, "Error reading grid size and tolerance\n");
        exit(EXIT_FAILURE);
    }

    initialize_grid(grid, fixed, *n);

    float value;
    int x1, y1, x2, y2;
    while (fscanf(file, "%f %d %d %d %d", &value, &x1, &y1, &x2, &y2) == 5) {
        for (int y = y1; y <= y2; ++y) {
            for (int x = x1; x <= x2; ++x) {
                if (x >= 0 && x < *n && y >= 0 && y < *n) {
                    grid[y * (*n) + x] = value;
                    fixed[y * (*n) + x] = true;
                }
            }
        }
    }

    fclose(file);
}

// Function to save results for visualization
void save_results(float *grid, int n, const char *filename) {
    // Create output filename
    char output_filename[256];
    snprintf(output_filename, sizeof(output_filename), "%s.dat", filename);
    
    FILE *file = fopen(output_filename, "w");
    if (!file) {
        fprintf(stderr, "Error creating output file\n");
        return;
    }
    
    // Write data in a format suitable for gnuplot
    for (int y = 0; y < n; ++y) {
        for (int x = 0; x < n; ++x) {
            fprintf(file, "%d %d %f\n", x, y, grid[y * n + x]);
        }
        fprintf(file, "\n"); // Add empty line for gnuplot 3D plotting
    }
    
    fclose(file);
    printf("Results saved to %s\n", output_filename);
}

// Main function
int main(int argc, char **argv) {
    int iterations_per_check = 1;
    bool show_timing = false;
    const char *filename = NULL;

    // Parse command-line arguments
    for (int i = 1; i < argc; ++i) {
        if (strcmp(argv[i], "-T") == 0) {
            show_timing = true;
        } else if (strcmp(argv[i], "-i") == 0 && i + 1 < argc) {
            iterations_per_check = atoi(argv[++i]);
        } else {
            filename = argv[i];
        }
    }

    if (!filename) {
        fprintf(stderr, "Usage: %s [-T] [-i m] filename\n", argv[0]);
        exit(EXIT_FAILURE);
    }

    // Read input file
    int n;
    float tolerance;
    float *grid;
    bool *fixed;
    
    // Allocate memory for grid and fixed flags
    grid = (float *)malloc(1024 * 1024 * sizeof(float)); // Max size
    fixed = (bool *)malloc(1024 * 1024 * sizeof(bool));
    
    if (!grid || !fixed) {
        fprintf(stderr, "Failed to allocate memory\n");
        exit(EXIT_FAILURE);
    }
    
    // Read the file and get the actual grid size
    read_input_file(filename, &n, &tolerance, grid, fixed);
    printf("Grid size: %d x %d, Tolerance: %g\n", n, n, tolerance);
    
    // Device memory
    float *device_grid1, *device_grid2, *device_delta;
    bool *device_fixed;
    
    // Allocate device memory
    hipMalloc(&device_grid1, n * n * sizeof(float));
    hipMalloc(&device_grid2, n * n * sizeof(float));
    hipMalloc(&device_fixed, n * n * sizeof(bool));
    hipMalloc(&device_delta, sizeof(float));
    
    // Copy data to device
    hipMemcpy(device_grid1, grid, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_grid2, grid, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_fixed, fixed, n * n * sizeof(bool), hipMemcpyHostToDevice);
    
    // Setup for computation
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((n + BLOCK_SIZE - 1) / BLOCK_SIZE, (n + BLOCK_SIZE - 1) / BLOCK_SIZE);
    
    // For timing
    hipEvent_t start, stop;
    float elapsed_time = 0.0f;
    
    if (show_timing) {
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
    }
    
    // Main computation loop
    int total_iterations = 0;
    float max_delta = tolerance * 2; // Start with a value greater than tolerance
    bool converged = false;
    
    while (!converged) {
        // Reset delta before reduction
        float zero = 0.0f;
        hipMemcpy(device_delta, &zero, sizeof(float), hipMemcpyHostToDevice);
        
        // Run computation kernel iterations_per_check times
        for (int i = 0; i < iterations_per_check; i++) {
            jacobi_compute_kernel<<<gridSize, blockSize>>>(device_grid1, device_grid2, n, device_fixed);
            hipDeviceSynchronize();
            
            // Swap grids for next iteration
            float *temp = device_grid1;
            device_grid1 = device_grid2;
            device_grid2 = temp;
            
            total_iterations++;
        }
        
        // Check for convergence
        reduction_kernel<<<gridSize, blockSize>>>(device_grid2, device_grid1, device_delta, n);
        hipDeviceSynchronize();
        
        hipMemcpy(&max_delta, device_delta, sizeof(float), hipMemcpyDeviceToHost);
        // Need to convert float bits representation back to float value
        max_delta = *(float*)&max_delta;
        
        printf("Iteration %d, Max delta: %g\n", total_iterations, max_delta);
        
        if (max_delta < tolerance) {
            converged = true;
            printf("Converged after %d iterations\n", total_iterations);
        }
    }
    
    // Report timing if requested
    if (show_timing) {
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed_time, start, stop);
        
        printf("\nTiming Results:\n");
        printf("Total computation time: %.2f ms\n", elapsed_time);
        printf("Average time per iteration: %.4f ms\n", elapsed_time / total_iterations);
        
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }
    
    // Copy results back to host
    hipMemcpy(grid, device_grid1, n * n * sizeof(float), hipMemcpyDeviceToHost);
    
    // Save results for visualization
    save_results(grid, n, filename);
    
    // Clean up
    free(grid);
    free(fixed);
    hipFree(device_grid1);
    hipFree(device_grid2);
    hipFree(device_fixed);
    hipFree(device_delta);
    
    return 0;
}